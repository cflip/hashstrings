
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define STRING_LENGTH 4

const int n = 26;
const int num_elements = pow(n, STRING_LENGTH);
const int threads_per_block = 256;

__device__ long compute_hash_code(const char* str, int n)
{
	long result = 0;
	for (int i = 0; i < n; i++)
		result = 31 * result + str[i];
	return result;
}

__global__ void find_from_generated_strings(long* hash_to_find, char* result_str,
		int total_elements)
{
	char str[STRING_LENGTH + 1];

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= total_elements)
		return;

	for (int j = 0; j < STRING_LENGTH; j++) {
		int string_index = STRING_LENGTH - (j + 1);
		int char_index = (i / (int)pow(n, j)) % n;
		str[string_index] = 'a' + char_index;
	}

	if (compute_hash_code(str, STRING_LENGTH) == *hash_to_find) {
		memcpy(result_str, str, STRING_LENGTH);
		return;
	}
}

int main()
{
	long hash_to_find = 3446974;
	char result[STRING_LENGTH + 1];

	long* to_find_on_device;
	char* result_string_on_device;

	hipMalloc(&to_find_on_device, sizeof(long));
	hipMalloc(&result_string_on_device, STRING_LENGTH + 1);

	hipMemcpy(to_find_on_device, &hash_to_find, sizeof(int),
			hipMemcpyHostToDevice);

	const int num_blocks = (num_elements + threads_per_block - 1) /
		threads_per_block;
	printf("We have %d items to compute\n", num_elements);
	printf("Running with %d blocks and %d threads per block\n", num_blocks,
			threads_per_block);
	find_from_generated_strings<<<num_blocks, threads_per_block>>>(to_find_on_device,
			result_string_on_device, num_elements);

	hipMemcpy(result, result_string_on_device, STRING_LENGTH + 1, hipMemcpyDeviceToHost);

	printf("%s\n", result);

	hipFree(to_find_on_device);
	hipFree(result_string_on_device);

	return 0;
}
